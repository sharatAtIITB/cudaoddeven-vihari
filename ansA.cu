#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include "utils.h"

extern __device__ __host__ inline void swap(int a[], int i, int j);

__global__ void _ansA(int* data, int n, int* sorted, int odd){
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid%2 != odd)
    return;
  if (tid<n && data[tid]>data[tid+1]){
    swap(data, tid, tid+1);
    *sorted = 0;
  }
}

void ansA(int* data, int nsize){
  int num_blocks = nsize;
  int threads_per_block = 1;
  int* dd;
  int is_sorted = 1;
  int *dsorted;
  hipMalloc(&dsorted, 1 * sizeof(int));
  hipMemset(dsorted, 1, sizeof(int));
  checkErrors("Failed to set");

  hipMalloc(&dd, nsize * sizeof(int));
  checkErrors("Failed to allocate");
  hipMemcpy(dd, data, nsize * sizeof(int), hipMemcpyHostToDevice);
  checkErrors("Failed to copy");

  do{
    hipMemset(dsorted, 1, sizeof(int));
    _ansA <<< num_blocks, threads_per_block>>> (dd, nsize, dsorted, 0);
    checkErrors("Could not make kernel launch");
    _ansA <<< num_blocks, threads_per_block>>> (dd, nsize, dsorted, 1);
    checkErrors("Could not make kernel launch");
    hipMemcpy(&is_sorted, dsorted, sizeof(int), hipMemcpyDeviceToHost);
    checkErrors("Failed to copy from device");
  }while(is_sorted==0);
  hipMemcpy(data, dd, sizeof(int)*nsize, hipMemcpyDeviceToHost);
}
